//testestest

#include "include/allocate.h"
#include <time.h>
#include <sys/time.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>

double *d_A, *d_B, *d_C;

double wallclock(void)
{
  struct timeval tv;
  struct timezone tz;
  double t;

  gettimeofday(&tv, &tz);

  t = (double)tv.tv_sec*1000;
  t += ((double)tv.tv_usec)/1000.0;

  return t;
}

int main(){

  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  double start, end;

  Init();

//cuda memory set
  hipMalloc((void**)&d_A, m*k*sizeof(double));
  hipMalloc((void**)&d_B, k*n*sizeof(double));
  cudaStat = hipMalloc((void**)&d_C, m*n*sizeof(double));

//data Host to Device
  hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, k*n*sizeof(double), hipMemcpyHostToDevice);

//handle
//  hipblasStatus_t hipblasCreate(hipblasHandle_t *handle);

  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
  printf ("CUBLAS initialization failed\n");
  return EXIT_FAILURE;
  }

  stat = hipblasSetMatrix (m, n, sizeof(double), C, m, d_C, m);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
  printf ("data download failed");
  hipFree (d_C);
  hipblasDestroy(handle);
  return EXIT_FAILURE;
  }
  start = wallclock();
  hipblasDgemm(  handle, 
		HIPBLAS_OP_N, 
		HIPBLAS_OP_N, 
		m, 
		n, 
		k, 
		&alpha, 
		d_A, 
		m, 
		d_B, 
		k, 
		&beta, 
		d_C, 
		n);
  hipDeviceSynchronize();
  end = wallclock();
  double time = (end - start) * 1e-3;

//data Device to Host
  hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

  double SIZE = (double)m*n*k;

  printf("Elapsed time is = %f seconds. \n\n", time);
  printf("GPU result : %f GFLOPS \n\n", SIZE / (end - start) * 1e-6);

  printf("%lf\t", C[0]);

  printf("\n");
  free(A), free(B), free(C);
  hipFree(d_A), hipFree(d_B), hipFree(d_C);
  hipblasDestroy(handle);

}

